#include "hip/hip_runtime.h"
#include<stdio.h>
#include<math.h>  // for ceil()
#include<stdexcept>

#include"tools.h"
#include"common.h"


// CUDA debugging
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define gpuKernelExecErrChk() { gpuCheckKernelExecutionError( __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if( abort )
			exit(code);
	}
}

inline void gpuCheckKernelExecutionError( const char *file, int line)
{
	/**
	  Check for invalid launch argument, then force the host to wait
	  until the kernel stops and checks for an execution error.
	  The synchronisation can be eliminated if there is a subsequent blocking
	  API call like cudaMemcopy. In this case the hipMemcpy call can return
	  either errors which occurred during the kernel execution or those from
	  the memory copy itself. This can be confusing for the beginner, so it is
	  recommended to use explicit synchronisation after a kernel launch during
	  debugging to make it easier to understand where problems might be arising.
	 */
	gpuAssert( hipPeekAtLastError(), file, line);
	gpuAssert( hipDeviceSynchronize(), file, line);	
}

//---------------------------------------------------------

// STUDENTS BEGIN

__global__ 
void kernelSpmvCSR(uint rowsNbr, const float *values, const uint *col_ind, const uint *row_ptr, const float *v, float *y)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	
	// Matrice:values[],col_ind[],row_ptr[]
	// Vecteur:v[]
	
	int row_beg = row_ptr[x];
	int row_end = row_ptr[x+1];
	
	// y[x] est dans la mémoire globale, on crée une variable dans la mémoire locale pour gagner du temps => utilisation d'un registre du GPU
	float dot = y[x];
	for (int i = row_beg; i < row_end; i++)
	{
		dot += values[i] * v[col_ind[i]];
	}
	y[x] = dot;
}

// STUDENTS END


//---------------------------------------------------------

/**
  Compute MxV on GPU. CSR method.
  A reference result can be passed to check that the computation is ok.
*/
Matrix* gpuSpmvCSR(const MatrixCSR *m, const Matrix *v, const Matrix *reference = NULL)
{
	const char *name = "CSR method on GPU";
	double gpuComputeTime = 0.0; // time measurement
	double gpuRunTime = 0.0; // time measurement

	if(m->w != v->h)
		throw std::runtime_error("Failed to multiply matrices, size mismatch.");
	if(v->w != 1)
		throw std::runtime_error("Failed to multiply matrices, vector size mismatch.");

	// output matrix size
	uint width = v->w;
	uint height = m->h;
	Matrix *mv = createMatrix(width, height);

	// data size
	uint valuesSizeInBytes = m->nzNbr * sizeof(float);
	uint col_indSizeInBytes = m->nzNbr * sizeof(uint);
	uint row_ptrSizeInBytes = (m->h + 1) * sizeof(uint);
	uint vSizeInBytes = (v->h) * sizeof(float);
	uint mvSizeInBytes = (m->h) * sizeof(float);

	// make a dummy memory allocation to wake up NVIDIA driver
	// before starting time measurement
	int *gpuWakeUp;
	gpuErrchk( hipMalloc( (void**) &gpuWakeUp, 1) );

	// STUDENTS BEGIN
	
	// On va appeler la méthode kernelSpmvCSR mais d'abord il faut allouer les paramètres à passer
	
	float* valuesIn;
	gpuErrchk(hipMalloc((void**) &valuesIn, valuesSizeInBytes));
	
	uint* colIndIn;
	gpuErrchk(hipMalloc((void**) &colIndIn, col_indSizeInBytes));
	
	uint* rowPtrIn;
	gpuErrchk(hipMalloc((void**) &rowPtrIn, row_ptrSizeInBytes));
	
	float* vIn;
	gpuErrchk(hipMalloc((void**) &vIn, vSizeInBytes));
	
	float* mvOut;
	gpuErrchk(hipMalloc((void**) &mvOut, mvSizeInBytes));
	
	// transfer data from CPU memory to GPU memory
	top(0); // start time measurement
	gpuErrchk(hipMemcpy(valuesIn, m->data, valuesSizeInBytes, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(colIndIn, m->col_ind, col_indSizeInBytes, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(rowPtrIn, m->row_ptr, row_ptrSizeInBytes, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(vIn, v->data, vSizeInBytes, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(mvOut, mv->data, mvSizeInBytes, hipMemcpyHostToDevice));
	
	dim3 dimBlock(32);
	dim3 dimGrid((int) ceilf(m->h*1.0/dimBlock.x));
	top(1);
	
	// Kernel launch
	kernelSpmvCSR<<<dimGrid, dimBlock>>>(m->h, valuesIn, colIndIn, rowPtrIn, vIn, mvOut);
	
	gpuKernelExecErrChk();
	gpuComputeTime = top(1); // pure computation duration
	
	gpuErrchk(hipMemcpy(mv->data, mvOut, mvSizeInBytes, hipMemcpyDeviceToHost));
	gpuRunTime = top(0); // computation and memory transfert time
	
	// release CPU memory
	hipFree(valuesIn);
	hipFree(colIndIn);
	hipFree(rowPtrIn);
	hipFree(vIn);
	hipFree(mvOut);
	
	// STUDENTS END

	// check result, display run time if result is correct
	bool displayRunTime = true;
	if( reference )
	{
		if(! checkResult(name, reference, mv))
			displayRunTime = false;
	}

	if(displayRunTime)
		printf("%s: M(%dx%d)xV computed in %f ms (%f ms of pure computation).\n", name, m->w, m->h, gpuRunTime, gpuComputeTime);

	return mv;
}

//---------------------------------------------------------

// STUDENTS BEGIN

/*__global__ void kernelSpmvELL(uint rowsNbr, const float *values, const uint *col_ind, uint nzRowSz, const float *v, float *y)
{
	
}*/

// STUDENTS END

//---------------------------------------------------------

/**
  Compute MxV on GPU. ELL method.
  A reference result can be passed to check that the computation is ok.
*/
Matrix* gpuSpmvELL(const MatrixELL *m, const Matrix *v, const Matrix *reference = NULL)
{
	const char *name = "ELL method on GPU";
	double gpuComputeTime = 0.0; // time measurement
	double gpuRunTime = 0.0; // time measurement

	if(m->w != v->h)
		throw std::runtime_error("Failed to multiply matrices, size mismatch.");
	if(v->w != 1)
		throw std::runtime_error("Failed to multiply matrices, vector size mismatch.");

	// output matrix size
	uint width = v->w;
	uint height = m->h;
	Matrix *mv = createMatrix(width, height);

	// data size
	uint valuesSizeInBytes = m->nzRowSz * m->h * sizeof(float);
	uint col_indSizeInBytes = m->nzRowSz * m->h * sizeof(uint);
	uint vSizeInBytes = (v->h) * sizeof(float);
	uint mvSizeInBytes = (m->h) * sizeof(float);

	// make a dummy memory allocation to wake up NVIDIA driver
	// before starting time measurement
	int *gpuWakeUp;
	gpuErrchk( hipMalloc( (void**) &gpuWakeUp, 1) );

	// STUDENTS BEGIN

	//TODO
	
	// STUDENTS END

	// check result, display run time if result is correct
	bool displayRunTime = true;
	if( reference )
	{
		if(! checkResult(name, reference, mv))
			displayRunTime = false;
	}

	if(displayRunTime)
		printf("%s: M(%dx%d)xV computed in %f ms (%f ms of pure computation).\n", name, m->w, m->h, gpuRunTime, gpuComputeTime);

	return mv;
}

//---------------------------------------------------------

// STUDENTS BEGIN

//kernelSpmvCSRVect(uint rowsNbr, const float *values, const uint *col_ind, const uint *row_ptr, const float *v, float *y)
//TODO

// STUDENTS END

//---------------------------------------------------------

/**
  Compute MxV on GPU. CSR-Vect method.
  A reference result can be passed to check that the computation is ok.
*/
Matrix* gpuSpmvCSRVect(const MatrixCSR *m, const Matrix *v, const Matrix *reference = NULL)
{
	const char *name = "CSR-Vect method on GPU";
	double gpuComputeTime = 0.0; // time measurement
	double gpuRunTime = 0.0; // time measurement

	if(m->w != v->h)
		throw std::runtime_error("Failed to multiply matrices, size mismatch.");
	if(v->w != 1)
		throw std::runtime_error("Failed to multiply matrices, vector size mismatch.");

	// output matrix size
	uint width = v->w;
	uint height = m->h;
	Matrix *mv = createMatrix(width, height);

	// data size
	uint valuesSizeInBytes = m->nzNbr * sizeof(float);
	uint col_indSizeInBytes = m->nzNbr * sizeof(uint);
	uint row_ptrSizeInBytes = (m->h + 1) * sizeof(uint);
	uint vSizeInBytes = (v->h) * sizeof(float);
	uint mvSizeInBytes = (m->h) * sizeof(float);

	// make a dummy memory allocation to wake up NVIDIA driver
	// before starting time measurement
	int *gpuWakeUp;
	gpuErrchk( hipMalloc( (void**) &gpuWakeUp, 1) );

	// STUDENTS BEGIN

	//TODO
	
	// STUDENTS END

	// check result, display run time if result is correct
	bool displayRunTime = true;
	if( reference )
	{
		if(! checkResult(name, reference, mv))
			displayRunTime = false;
	}

	if(displayRunTime)
		printf("%s: M(%dx%d)xV computed in %f ms (%f ms of pure computation).\n", name, m->w, m->h, gpuRunTime, gpuComputeTime);

	return mv;
}
